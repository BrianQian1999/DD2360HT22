

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

#define DataType double

// CPU timer
struct timeval t_start, t_end;
void cputimer_start(){
    gettimeofday(&t_start, 0);
}
void cputimer_stop(const char* info){
    gettimeofday(&t_end, 0);
    double time = (1000000.0*(t_end.tv_sec-t_start.tv_sec) + t_end.tv_usec-t_start.tv_usec);
    printf("Timing - %s. \t\tElasped %.0f microseconds \n", info, time);
}

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

int main(int argc, char **argv) {
    int inputLength;

    DataType *hostInput1;
    DataType *hostInput2;
    DataType *hostOutput;
    DataType *resultRef;
    DataType *deviceInput1;
    DataType *deviceInput2;
    DataType *deviceOutput;

    //@@ Insert code below to read in inputLength from args
    inputLength = atoi(argv[1]);
    printf("The input length is %d\n", inputLength);

    // @@ Insert code below to allocate Host memory for input and output
    // To launch kernel on different stream, Pinned-memory must be used. Why? 
    hipHostMalloc((void**)&hostInput1, inputLength * sizeof(DataType), hipHostMallocDefault);
    hipHostMalloc((void**)&hostInput2, inputLength * sizeof(DataType), hipHostMallocDefault);
    hipHostMalloc((void**)&hostOutput, inputLength * sizeof(DataType), hipHostMallocDefault);

    //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = (double) rand() / RAND_MAX;
        hostInput2[i] = (double) rand() / RAND_MAX;
    }  
  
    //@@ Insert code below to allocate GPU memory here
    hipMalloc((void**) &deviceInput1, sizeof(DataType) * inputLength);
    hipMalloc((void**) &deviceInput2, sizeof(DataType) * inputLength);
    hipMalloc((void**) &deviceOutput, sizeof(DataType) * inputLength);

    // Start CPU timer
    // cputimer_start();
    hipProfilerStart();

    hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(DataType), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(DataType), hipMemcpyHostToDevice);
    
    // Launch the vecAdd kernel on the GPU
    dim3 dimGrid(ceil(inputLength / 256));
    dim3 dimBlock(256);
    vecAdd<<<dimGrid, dimBlock, 0>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(DataType), hipMemcpyDeviceToHost);

    // Wait until all issued CUDA calls are complete, i.e., hostOutput is correct
    hipDeviceSynchronize();

    // Stop CPU timer
    // cputimer_stop("Non-streamed vecAdd execution time (H2D + kernel + D2H)");
    hipProfilerStop();

    //@@ Insert code below to compare the output with the reference
    bool correct = true;
    resultRef = (DataType*)malloc(sizeof(DataType) * inputLength);
    for (int i = 0; i < inputLength; i++) {
        resultRef[i] = hostInput1[i] + hostInput2[i];
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-8) {
            correct = false;
            break;
        }
    }
    printf("Result is %s\n", correct ? "CORRECT" : "INCORRECT");


    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    //@@ Free the CPU memory here
    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);
    free(resultRef);
    return 0;
}



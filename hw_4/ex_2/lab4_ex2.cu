

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

int main(int argc, char **argv) {
    int inputLength;
    int S_seg; // number of segments

    DataType *hostInput1;
    DataType *hostInput2;
    DataType *hostOutput;
    DataType *resultRef;
    DataType *deviceInput1;
    DataType *deviceInput2;
    DataType *deviceOutput;

    //@@ Insert code below to read in inputLength from args
    inputLength = atoi(argv[1]);
    printf("The input length is %d\n", inputLength);
    S_seg = atoi(argv[2]);
    printf("The length of each segment is %d\n", S_seg);
    const int num_segments = ceil(inputLength / S_seg);
    printf("The number of segments is %d\n", num_segments);

    // @@ Insert code below to allocate Host memory for input and output
    // To launch kernel on different stream, Pinned-memory must be used. Why? 
    hipMalloc((void**)&hostInput1, inputLength * sizeof(DataType));
    hipMalloc((void**)&hostInput2, inputLength * sizeof(DataType));
    hipMalloc((void**)&hostOutput, inputLength * sizeof(DataType));

    //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = (double) rand() / RAND_MAX;
        hostInput2[i] = (double) rand() / RAND_MAX;
    }  
  
    //@@ Insert code below to allocate GPU memory here
    hipMalloc((void**) &deviceInput1, sizeof(DataType) * inputLength);
    hipMalloc((void**) &deviceInput2, sizeof(DataType) * inputLength);
    hipMalloc((void**) &deviceOutput, sizeof(DataType) * inputLength);

    // Create CUDA streams
    hipStream_t streams[num_segments]; 
    for(int i = 0; i < num_segments; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Divide the input vector into segments and copy each segment asynchronously to the GPU
    for(int i = 0; i < num_segments; i++) {
        int offset = i * S_seg;
        hipMemcpyAsync(deviceInput1 + offset, hostInput1 + offset, S_seg * sizeof(DataType), 
                        hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(deviceInput2 + offset, hostInput2 + offset, S_seg * sizeof(DataType), 
                        hipMemcpyHostToDevice, streams[i]); 
    }
    
    // Launch the vecAdd kernel on the GPU
    for(int i = 0; i < num_segments; i++) {
        int offset = i * S_seg;
        dim3 dimGrid(ceil(S_seg / 256));
        dim3 dimBlock(256);
        vecAdd<<<dimGrid, dimBlock, 0, streams[i]>>>
            (deviceInput1 + offset, deviceInput2 + offset, deviceOutput + offset, S_seg);
    }

    // Copy the output segments back to the host asynchronously
    for(int i = 0; i < num_segments; i++) {
        int offset = i * S_seg;
        hipMemcpyAsync(hostOutput + offset, deviceOutput + offset, S_seg * sizeof(DataType), 
                        hipMemcpyDeviceToHost, streams[i]);
    }

    // Wait until all issued CUDA calls are complete, i.e., hostOutput is correct
    hipDeviceSynchronize();

    //@@ Insert code below to compare the output with the reference
    bool correct = true;
    resultRef = (DataType*)malloc(sizeof(DataType) * inputLength);
    for (int i = 0; i < inputLength; i++) {
        resultRef[i] = hostInput1[i] + hostInput2[i];
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-8) {
            correct = false;
            break;
        }
    }
    printf("Result is %s\n", correct ? "CORRECT" : "INCORRECT");


    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    //@@ Free the CPU memory here
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
    free(resultRef);
    return 0;
}


